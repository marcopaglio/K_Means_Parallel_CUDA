#include "hip/hip_runtime.h"
/*
 * KMeans.cpp
 *
 *  Created on: 8 feb 2021
 *      Author: marco
 */

#include "KMeans.h"
#include "Point.h"
#include <cmath>

#include <cstdio>
#include <cassert>
#include <iostream>

static void CheckCudaErrorAux(const char *, unsigned, const char *,
		hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

#define TOLERANCE 0.005
__device__ unsigned int d_newCentroidIndex;
__device__ float d_maxMinDistance;
__device__ unsigned int cudaLock;

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "("
			<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}

__host__ SetOfPoints* kMeans(unsigned int k, const SetOfPoints& data) noexcept(false) {
    Point* pointList = data.pointList;
    if(pointList == nullptr) {
        throw invalid_argument("Clusters can't be null");
    }
    int numPoints = data.sizeList;
    if (numPoints < k) {
        throw length_error("There aren't enough points for k = " + to_string(k));
    }

    SetOfPoints* clusters = SetOfPoints_new((Point *) calloc(k, sizeof (Point)), k);
    if (k == 1) {
        setAttributes(&(clusters[0]), pointList, numPoints);
        return clusters;
    }

    Point* centroids = initialCentroids(k, data);
    unsigned int* clusterization = (unsigned int *) calloc(numPoints, sizeof(unsigned int));
    Point* oldCentroids;
    unsigned int* clusterSize = (unsigned int *) calloc(k, sizeof(unsigned int));
    unsigned int dim = data.pointList[0].dimension;
    float* sum = (float *) calloc(k * dim, sizeof(float));

    bool stop = false;
    while (!stop) {
        updateClusters(k, data, centroids, clusterization, clusterSize, sum);
        oldCentroids = centroids;
        centroids = updateCentroids(k, dim, clusterSize, sum);

        if(checkStop(k, oldCentroids, centroids)) {
            stop = true;
        }

        for (int z = 0; z < k; z++) {
            free(oldCentroids[z].coordinates);
        }
        free(oldCentroids);
    }
    for (int z = 0; z < k; z++) {
        free(centroids[z].coordinates);
    }
    free(centroids);
    free(sum);

    unsigned int clusterIndex [k];
	for (unsigned int c = 0; c < k; c++) {
		setAttributes(&(clusters[c]), (Point *) calloc(clusterSize[c], sizeof(Point)), clusterSize[c]);
		clusterIndex[c] = 0;
	}
	for (unsigned int p = 0; p < numPoints; p++) {
		insertPoint(&(clusters[clusterization[p]]), pointList[p], clusterIndex[clusterization[p]]);
		clusterIndex[clusterization[p]]++;
	}

    free(clusterSize);
    free(clusterization);
    return clusters;
}

__host__ bool checkStop(unsigned int k, const Point* oldCentroids, const Point* newCentroids) {
    unsigned int dimension = oldCentroids[0].dimension;
    for (unsigned int c = 0; c < k; c++) {
        float* oldCoordinates = oldCentroids[c].coordinates;
        float* newCoordinates = newCentroids[c].coordinates;
        for (unsigned int d = 0; d < dimension; d++) {
            if (abs(oldCoordinates[d] - newCoordinates[d]) > TOLERANCE) {
                return false;
            }
        }
    }
    return true;
}

__host__ Point* updateCentroids(unsigned int k, unsigned int dim, unsigned int* clusterSize, float* sum) {
    string fakeMeta;

    Point* centroids = (Point*) calloc(k, sizeof(Point));
    for (unsigned int c = 0; c < k; c++) {
        float* coordinates = (float*) calloc(dim, sizeof(float));
        for (unsigned int d = 0; d < dim; d++) {
            coordinates[d] = sum[c * dim + d] / (float) clusterSize[c];
        }
        setAttributes(&(centroids[c]), coordinates, dim, fakeMeta);
    }

    return centroids;
}

__global__ void updateClustersKernel(unsigned int k, Point* d_pointList, unsigned int size, unsigned int from, Point* c_centroids, unsigned int* d_clusterization, unsigned int* d_clusterSize, float* d_sum) {
	unsigned int p = blockIdx.x * blockDim.x + threadIdx.x;

	if (p < size) {
		unsigned int dim = d_pointList[0].dimension;
		float minDistance = INFINITY;
        unsigned int nearestCentroid;
        float distance;

        for (unsigned int c = 0; c < k; c++) {
            distance = getDistance(d_pointList[p], c_centroids[c]);
            if (distance < minDistance) {
                minDistance = distance;
                nearestCentroid = c;
            }
        }
        d_clusterization[from + p] = nearestCentroid;
        atomicAdd(&(d_clusterSize[nearestCentroid]), 1);
        for (unsigned int d = 0; d < dim; d++) {
        	atomicAdd(&(d_sum[nearestCentroid * dim + d]), getCoordinateAt(d_pointList[p], d));
        }
	}
}

__host__ void updateClusters(unsigned int k, const SetOfPoints& data, Point* centroids, unsigned int* clusterization, unsigned int* clusterSize, float* sum) {
    unsigned int numPoints = data.sizeList;
	unsigned int dim = data.pointList[0].dimension;
	for (unsigned int c = 0; c < k; c++) {
        clusterSize[c] = 0;
        for (unsigned int d = 0; d < dim; d++) {
        	sum[c * dim + d] = 0;
        }
    }

	unsigned int* d_clusterSize;
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_clusterSize, k * sizeof(unsigned int)));
	CUDA_CHECK_RETURN(hipMemcpy((void*)d_clusterSize, (void*)clusterSize, k * sizeof(unsigned int), hipMemcpyHostToDevice));
	unsigned int* d_clusterization;
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_clusterization, numPoints * sizeof(unsigned int)));
	CUDA_CHECK_RETURN(hipMemcpy((void*)d_clusterization, (void*)clusterization, numPoints * sizeof(unsigned int), hipMemcpyHostToDevice));
	float* d_sum;
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_sum, k * dim * sizeof(float)));
	CUDA_CHECK_RETURN(hipMemcpy((void*)d_sum, (void*)sum, k * dim * sizeof(float), hipMemcpyHostToDevice));
	float* d_copyCoordinates;
	Point* c_centroids;
	CUDA_CHECK_RETURN(hipMalloc((void**)&c_centroids, k * sizeof(Point)));
	CUDA_CHECK_RETURN(hipMemcpy((void*)c_centroids, (void*)centroids, k * sizeof(Point), hipMemcpyHostToDevice));
	for (unsigned int c = 0; c < k; c++) {
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_copyCoordinates, dim * sizeof(float)));
		CUDA_CHECK_RETURN(hipMemcpy((void*)d_copyCoordinates,
				(void*)centroids[c].coordinates, dim * sizeof(float), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy((void*)&((c_centroids + c)->coordinates), (void*)&d_copyCoordinates,
				sizeof((c_centroids + c)->coordinates), hipMemcpyHostToDevice));
	}

	hipDeviceProp_t dev_prop;
	hipGetDeviceProperties(&dev_prop, 0);

	unsigned int blockSize = 128;
	unsigned int gridSizeNecessary = ceil(data.sizeList / (float) blockSize);
    unsigned int gridSize = dev_prop.maxGridSize[0];

    Point* d_pointList;
    unsigned int from = 0;
    unsigned int size;
    unsigned int p;

    while (gridSizeNecessary > 0) {
		if (gridSizeNecessary > dev_prop.maxGridSize[0]) { //maxGridSize = 2 miliardi, sicuramente troppo, vedi FIXME successivo
			size = gridSize * blockSize;
		} else {
			size = data.sizeList - from;
			gridSize = gridSizeNecessary;
		}
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_pointList, size * sizeof(Point))); //FIXME returned out of memory because too much great
		CUDA_CHECK_RETURN(hipMemcpy((void*)d_pointList, (void*)&data.pointList[from],
				size * sizeof(Point), hipMemcpyHostToDevice));
		for (p = 0; p < size; p++) {
			CUDA_CHECK_RETURN(hipMalloc((void**)&d_copyCoordinates, dim * sizeof(float)));
			CUDA_CHECK_RETURN(hipMemcpy((void*)d_copyCoordinates,
					(void*)data.pointList[from + p].coordinates, dim * sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK_RETURN(hipMemcpy((void*)&((d_pointList + p)->coordinates),
					(void*)&d_copyCoordinates, sizeof((d_pointList + p)->coordinates), hipMemcpyHostToDevice));
		}

		//CALL kernel
		updateClustersKernel<<<gridSize, blockSize>>>(k, d_pointList, size, from, c_centroids, d_clusterization, d_clusterSize, d_sum);

		from += size;
		gridSizeNecessary -= gridSize;
		hipDeviceSynchronize();

		for (p = 0; p < size; p++) {
			CUDA_CHECK_RETURN(hipMemcpy((void*)&d_copyCoordinates, (void*)&((d_pointList + p)->coordinates),
							sizeof((d_pointList + p)->coordinates), hipMemcpyDeviceToHost));
			CUDA_CHECK_RETURN(hipFree(d_copyCoordinates));
		}
		CUDA_CHECK_RETURN(hipFree(d_pointList));
	}

    CUDA_CHECK_RETURN(hipMemcpy((void*)sum, (void*)d_sum, k * dim * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipFree(d_sum));
    CUDA_CHECK_RETURN(hipMemcpy((void*)clusterSize, (void*)d_clusterSize, k * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipFree(d_clusterSize));
    CUDA_CHECK_RETURN(hipMemcpy((void*)clusterization, (void*)d_clusterization, numPoints * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipFree(d_clusterization));
    for (unsigned int c = 0; c < k; c++) {
		CUDA_CHECK_RETURN(hipMemcpy((void*)&d_copyCoordinates, (void*)&((c_centroids + c)->coordinates),
				sizeof((c_centroids + c)->coordinates), hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipFree(d_copyCoordinates));
	}
	CUDA_CHECK_RETURN(hipFree(c_centroids));
}

__global__ void maxMinDistanceKernel(unsigned int i, Point* d_pointList, unsigned int size, unsigned int from, Point* c_centroids) {
	__shared__ float ds_maxMinDistances[128];
	__shared__ unsigned int ds_maxMinIndexes[128];

	unsigned p = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int t = threadIdx.x;

	if (p % 1000 == 0) {
		printf("K = %d, thread %d utilizza r: %f, g: %f, b: %f.\n\n", i, p, d_pointList[p].coordinates[0], d_pointList[p].coordinates[1], d_pointList[p].coordinates[2]);
	}
	if (p < size) {
		float minDistance = INFINITY;
		float distance;
		for (unsigned int j = 0; j < i; j++) {
			distance = getDistance(d_pointList[p], c_centroids[j]);
			if (distance < minDistance) {
				minDistance = distance;
			}
		}

		ds_maxMinDistances[t] = minDistance;
		ds_maxMinIndexes[t] = from + p;
	} else {
		ds_maxMinDistances[t] = 0;			//0 will never be caught
		ds_maxMinIndexes[t] = from + size;	//from+size is out of range
	}

	for (unsigned int stride = blockDim.x / 2; stride >= 1; stride /= 2) {
		__syncthreads();

		if (t >= stride) {
			return; //TESTED with k=10 less than if(t < stride) {} (TOT: 82s vs 83s)
		} else {
		//if(t < stride){
			if (ds_maxMinDistances[t] < ds_maxMinDistances[t + stride]) {
				ds_maxMinDistances[t] = ds_maxMinDistances[t + stride];
				ds_maxMinIndexes[t] = ds_maxMinIndexes[t + stride];
			}
			if (stride == 1) {
				bool blocked = true;
				while (blocked) {
					if (0 == atomicCAS(&cudaLock, 0, 1)) {
						if (d_maxMinDistance < ds_maxMinDistances[0]) {
							atomicExch(&d_maxMinDistance, ds_maxMinDistances[0]);
							atomicExch(&d_newCentroidIndex, ds_maxMinIndexes[0]);
						}
						atomicExch(&cudaLock, 0);
						blocked = false;
					}
				}
			}
		}
	}
}

__host__ Point* initialCentroids(unsigned int k, const SetOfPoints& data) {
    if (data.sizeList == k) {
        return data.pointList;
    }
    Point* centroids = (Point*) calloc(k, sizeof(Point));
	Point* c_centroids;
	CUDA_CHECK_RETURN(hipMalloc((void**)&c_centroids, k * sizeof(Point)));

	unsigned int dim = data.pointList[0].dimension;
	float* d_copyCoordinates;

    int firstIndex = 0;
    copyPoint(&(centroids[0]), &(data.pointList[firstIndex]));

    /**** NEXT CODE IS USED TO VERIFY INIT IS DONE
    std::cout << "Centroide 0-esimo" << std::endl;
    std::cout << "dim: " << centroids[0].dimension << std::endl;
    std::cout << "meta " << centroids[0].metadata << std::endl;
    for(int i=0; i<dim; i++) {
		std::cout << i << " " << centroids[0].coordinates[i] << std::endl;
	}
	*****/

    //Every deep copy in device memory consists in 4 steps:
    //1) init struct in device memory
    CUDA_CHECK_RETURN(hipMemcpy((void*)&(c_centroids[0]),
        				(void*)&centroids[0], sizeof(Point), hipMemcpyHostToDevice));
    //2) allocate array in device memory
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_copyCoordinates, dim * sizeof(float)));
    //3) init array in device memory
    CUDA_CHECK_RETURN(hipMemcpy((void*)d_copyCoordinates,
    		(void*)centroids[0].coordinates, dim * sizeof(float), hipMemcpyHostToDevice));

    /**** NEXT CODE IS USED TO VERIFY COPY IS DONE
    memset(centroids[0].coordinates, 0, dim * sizeof(float));
    hipMemcpy(&centroids[0], &c_centroids[0], sizeof(Point), hipMemcpyDeviceToHost);
    hipMemcpy(centroids[0].coordinates, d_copyCoordinates, dim * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "dim " << centroids[0].dimension << std::endl;
    std::cout << "meta " << centroids[0].metadata << std::endl;
    for(int i=0; i<dim; i++) {
        std::cout << i << " " << centroids[0].coordinates[i] << std::endl;
    }
    *****/

    //4) move device array pointer to device struct field
    CUDA_CHECK_RETURN(hipMemcpy((void*)&((c_centroids + 0)->coordinates), (void*)&d_copyCoordinates,
    		sizeof((c_centroids + 0)->coordinates), hipMemcpyHostToDevice));

    hipDeviceProp_t dev_prop;
    hipGetDeviceProperties(&dev_prop, 0); //device = GPU ?? Sì

    unsigned int blockSize = 128;
    unsigned int gridSizeNecessary;

    unsigned int h_newCentroidIndex;
    float h_maxMinDistance;

    Point* d_pointList;
    unsigned int from;
    unsigned int size;
    unsigned int gridSize = dev_prop.maxGridSize[0];
    unsigned int p;

    for (unsigned int i = 1; i < k; i++) {
        h_maxMinDistance = 0;
        CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(d_maxMinDistance), &h_maxMinDistance, sizeof(float)));

        /**** NEXT CODE IS USED TO VERIFY COPY IS DONE
        CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&h_maxMinDistance, HIP_SYMBOL(d_maxMinDistance), sizeof(float)));
        std::cout << "init d_maxMinDistance at " << h_maxMinDistance << std::endl;
        ****/

        from = 0;
        gridSizeNecessary = ceil(data.sizeList / (float) blockSize); //need cast to float in order to have right value

        while (gridSizeNecessary > 0) {
        	if (gridSizeNecessary > dev_prop.maxGridSize[0]) { //maxGridSize = 2 miliardi, sicuramente troppo, vedi FIXME successivo
        		size = gridSize * blockSize;
        	} else {
        		size = data.sizeList - from;
        		gridSize = gridSizeNecessary;
        	}
        	CUDA_CHECK_RETURN(hipMalloc((void**)&d_pointList, size * sizeof(Point))); //FIXME returned out of memory because too much great
        	CUDA_CHECK_RETURN(hipMemcpy((void*)d_pointList, (void*)&data.pointList[from],
        			size * sizeof(Point), hipMemcpyHostToDevice));
        	for (p = 0; p < size; p++) {
        		CUDA_CHECK_RETURN(hipMalloc((void**)&d_copyCoordinates, dim * sizeof(float)));
        		CUDA_CHECK_RETURN(hipMemcpy((void*)d_copyCoordinates,
        				(void*)data.pointList[from + p].coordinates, dim * sizeof(float), hipMemcpyHostToDevice));
        		CUDA_CHECK_RETURN(hipMemcpy((void*)&((d_pointList + p)->coordinates),
        				(void*)&d_copyCoordinates, sizeof((d_pointList + p)->coordinates), hipMemcpyHostToDevice));

				/**** NEXT CODE IS USED TO VERIFY COPY IS DONE
				if (p % 1000 == 0) {
					std::cout << "Punto " << p << "-esimo: " << std::endl;
					for(int i=0; i<dim; i++) {
						std::cout << i << " " << data.pointList[from + p].coordinates[i] << std::endl;
					}
					memset(data.pointList[from + p].coordinates, 0, dim * sizeof(float));
					hipMemcpy(data.pointList[from + p].coordinates, d_copyCoordinates, dim * sizeof(float), hipMemcpyDeviceToHost);
					for(int i=0; i<dim; i++) {
						std::cout << i << " " << data.pointList[from + p].coordinates[i] << std::endl;
					}
				}
				****/
        	}

        	//CALL kernel
        	maxMinDistanceKernel<<<gridSize, blockSize>>>(i, d_pointList, size, from, c_centroids);

        	from += size;
        	gridSizeNecessary -= gridSize;
        	hipDeviceSynchronize();

        	/**** NEXT CODE IS USED TO VERIFY IF KERNEL HAS LAUNCHED ERRORS
        	std::string error = hipGetErrorString(hipPeekAtLastError());
        	std::cout << error << std::endl;
        	error = hipGetErrorString(hipDeviceSynchronize());
        	std::cout << error << std::endl;
        	****/

        	for (p = 0; p < size; p++) {
				//access to device pointer is not possible, so...
				//1) copy pointer of device pointer in pointer of host-defined device pointer
				CUDA_CHECK_RETURN(hipMemcpy((void*)&d_copyCoordinates, (void*)&((d_pointList + p)->coordinates),
								sizeof((d_pointList + p)->coordinates), hipMemcpyDeviceToHost));
				//2) free host defined device pointer
				CUDA_CHECK_RETURN(hipFree(d_copyCoordinates));
			}
			// This doesn't need to copy because is a host-defined device pointer
			CUDA_CHECK_RETURN(hipFree(d_pointList));
        }

        CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&h_newCentroidIndex, HIP_SYMBOL(d_newCentroidIndex), sizeof(unsigned int)));
        copyPoint(&(centroids[i]), &(data.pointList[h_newCentroidIndex]));

        /**** NEXT CODE IS USED TO VERIFY INIT IS DONE
        std::cout << "Centroide " << i << "-esimo" << std::endl;
        std::cout << "dim: " << centroids[i].dimension << std::endl;
        std::cout << "meta " << centroids[i].metadata << std::endl;
        for(int j=0; j<dim; j++) {
    		std::cout << j << " " << centroids[i].coordinates[j] << std::endl;
    	}
    	****/

        CUDA_CHECK_RETURN(hipMemcpy((void*)&c_centroids[i],
        		(void*)&centroids[i], sizeof(Point), hipMemcpyHostToDevice));
        CUDA_CHECK_RETURN(hipMalloc((void**)&d_copyCoordinates, dim * sizeof(float)));
        CUDA_CHECK_RETURN(hipMemcpy((void*)d_copyCoordinates,
        		(void*)centroids[i].coordinates, dim * sizeof(float), hipMemcpyHostToDevice));

        /**** NEXT CODE IS USED TO VERIFY COPY IS DONE
        memset(centroids[i].coordinates, 0, dim * sizeof(float));
        hipMemcpy(&centroids[i], &c_centroids[i], sizeof(Point), hipMemcpyDeviceToHost);
        hipMemcpy(centroids[i].coordinates, d_copyCoordinates, dim * sizeof(float), hipMemcpyDeviceToHost);

        std::cout << "dim " << centroids[i].dimension << std::endl;
        std::cout << "meta " << centroids[i].metadata << std::endl;
        for(int j=0; j<dim; j++) {
            std::cout << j << " " << centroids[i].coordinates[j] << std::endl;
        }
        ****/

        CUDA_CHECK_RETURN(hipMemcpy((void*)&((c_centroids + i)->coordinates), (void*)&d_copyCoordinates,
        		sizeof((c_centroids + i)->coordinates), hipMemcpyHostToDevice));
    }

    for (unsigned int c = 0; c < k; c++) {
    	CUDA_CHECK_RETURN(hipMemcpy((void*)&d_copyCoordinates, (void*)&((c_centroids + c)->coordinates),
    			sizeof((c_centroids + c)->coordinates), hipMemcpyDeviceToHost));
    	CUDA_CHECK_RETURN(hipFree(d_copyCoordinates));
    }
    CUDA_CHECK_RETURN(hipFree(c_centroids));

    return centroids;
}
